#include "hip/hip_runtime.h"

#include "gbkfit/models/model01/kernels_cuda_device.cuh"

namespace gbkfit {
namespace models {
namespace model01 {
namespace kernels_cuda_device {

__global__
void foo(float* out_velmap,
         float* out_sigmap,
         int data_size_x,
         int data_size_y)
{
    out_velmap[0] = 101;
}

} // namespace kernels_cuda_device
} // namespace model01
} // namespace models
} // namespace gbkfit
